#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void printThreadAndBlockIndices() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int idz = threadIdx.z + blockIdx.z * blockDim.z;
    printf("ThreadIdx (%d, %d, %d), BlockIdx (%d, %d, %d), Total ThreadIdx (%d, %d, %d)\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           idx, idy, idz);
}

int main() {
    dim3 block(8, 8, 4); 
    dim3 grid(4, 4, 4); 

    printThreadAndBlockIndices<<<grid, block>>>();
    hipDeviceSynchronize();

    hipError_t cuda_error = hipGetLastError();
    if (cuda_error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(cuda_error));
    }

    hipDeviceReset();
    return 0;
}
